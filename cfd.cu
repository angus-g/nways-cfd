#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <algorithm>

#include "arraymalloc.h"
#include "boundary.h"
#include "cfdio.h"
#include "jacobi.h"
#include "error.h"

int main(int argc, char **argv) {
  int printfreq = 1000; // output frequency
  double error, bnorm;
  double tolerance = 0; // tolerance for convergence. <=0 means do not check

  // main arrays
  double *psi, *zet;
  // temporary versions of main arrays
  double *psi_old, *psi_new;
  double *zet_old, *zet_new;

  // command line arguments
  int scalefactor, numiter;

  size_t sizef;

  double re; // Reynold's number - must be less than 3.7

  // simulation sizes
  int bbase = 10;
  int hbase = 15;
  int wbase = 5;
  int mbase = 32;
  int nbase = 32;

  int irrotational = 1, checkerr = 0;

  int m, n, b, h, w;
  int iter;
  int i, j;

  int device;

  double tstart, tstop, ttot, titer;

  // do we stop because of tolerance?
  if (tolerance > 0) {
    checkerr = 1;
  }

  // check command line parameters and parse them

  if (argc < 3 || argc > 4) {
    printf("Usage: cfd <scale> <numiter> [reynolds]\n");
    return 0;
  }

  scalefactor = atoi(argv[1]);
  numiter = atoi(argv[2]);

  if (argc == 4) {
    re = atof(argv[3]);
    irrotational = 0;
  } else {
    re = -1.0;
  }

  if (!checkerr) {
    printf("Scale Factor = %i, iterations = %i\n", scalefactor, numiter);
  } else {
    printf("Scale Factor = %i, iterations = %i, tolerance= %g\n", scalefactor,
           numiter, tolerance);
  }

  if (irrotational) {
    printf("Irrotational flow\n");
  } else {
    printf("Reynolds number = %f\n", re);
  }

  // Calculate b, h & w and m & n
  b = bbase * scalefactor;
  h = hbase * scalefactor;
  w = wbase * scalefactor;
  m = mbase * scalefactor;
  n = nbase * scalefactor;

  re = re / (double)scalefactor;

  printf("Running CFD on %d x %d grid in cuda\n", m, n);

  device = 0;
  HANDLE_ERROR(hipSetDevice(device));

  // allocate arrays

  sizef = (m + 2) * (n + 2) * sizeof(double);
  HANDLE_ERROR(hipHostAlloc((void **)&psi, sizef, hipHostMallocDefault));
  HANDLE_ERROR(hipMalloc((void **)&psi_old, sizef));
  HANDLE_ERROR(hipMalloc((void **)&psi_new, sizef));

  nvtxRangePush("Initialization");
  // zero the psi array
  for (i = 0; i < m + 2; i++) {
    for (j = 0; j < n + 2; j++) {
      psi[i * (m + 2) + j] = 0.0;
    }
  }
  nvtxRangePop(); // pop

  if (!irrotational) {
    // allocate arrays
    HANDLE_ERROR(hipHostAlloc((void **)&zet, sizef, hipHostMallocDefault));
    HANDLE_ERROR(hipMalloc((void **)&zet_old, sizef));
    HANDLE_ERROR(hipMalloc((void **)&zet_new, sizef));

    // zero the zeta array
    nvtxRangePush("Initialization");

    for (i = 0; i < m + 2; i++) {
      for (j = 0; j < n + 2; j++) {
        zet[i * (m + 2) + j] = 0.0;
      }
    }
    nvtxRangePop(); // pop for reading file
  }

  // set the psi boundary conditions
  nvtxRangePush("Boundary_PSI");

  boundarypsi(psi, m, n, b, h, w);
  nvtxRangePop(); // pop

  // compute normalisation factor for error

  bnorm = 0.0;
  nvtxRangePush("Compute_Normalization");

  for (i = 0; i < m + 2; i++) {
    for (j = 0; j < n + 2; j++) {
      bnorm += psi[i * (m + 2) + j] * psi[i * (m + 2) + j];
    }
  }
  nvtxRangePop(); // pop

  if (!irrotational) {
    // update zeta BCs that depend on psi
    boundaryzet(zet, psi, m, n);

    // update normalisation
    nvtxRangePush("Compute_Normalization");
    for (i = 0; i < m + 2; i++) {
      for (j = 0; j < n + 2; j++) {
        bnorm += zet[i * (m + 2) + j] * zet[i * (m + 2) + j];
      }
    }
    nvtxRangePop(); // pop
  }

  bnorm = sqrt(bnorm);

  // copy initialised field to device
  HANDLE_ERROR(hipMemcpy(psi_old, psi, sizef, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(psi_new, psi, sizef, hipMemcpyHostToDevice));

  // begin iterative Jacobi loop
  printf("\nStarting main loop...\n\n");

  tstart = gettime();
  nvtxRangePush("Overall_Iteration");

  dim3 dimBlock(32, 32);
  dim3 dimGrid(64, 64);

  for (iter = 1; iter <= numiter; iter++) {
    // calculate psi for next iteration
    nvtxRangePush("JacobiStep");
    if (irrotational) {
      jacobistep<<<dimGrid, dimBlock>>>(psi_new, psi_old, m, n);
      HANDLE_ERROR(hipPeekAtLastError());
    } else {
      jacobistepvort<<<dimGrid, dimBlock>>>(zet_new, psi_new, zet_old, psi_old, m, n, re);
    }
    nvtxRangePop(); // pop
    nvtxRangePush("Calculate_Error");
    // calculate current error if required

    if (checkerr || iter == numiter) {
      error = deltasq(dimGrid, dimBlock, psi_new, psi_old, m, n);

      if (!irrotational) {
        error += deltasq(dimGrid, dimBlock, zet_new, zet_old, m, n);
      }

      error = sqrt(error);
      error = error / bnorm;
    }
    nvtxRangePop(); // pop

    // quit early if we have reached required tolerance

    if (checkerr) {
      if (error < tolerance) {
        printf("Converged on iteration %d\n", iter);
        break;
      }
    }

    // copy back
    nvtxRangePush("Switch_Array");

    // swap old/new arrays
    //HANDLE_ERROR(hipMemcpy(psi_d, psitmp_d, sizef, hipMemcpyDeviceToDevice));
    //HANDLE_ERROR(hipPeekAtLastError());

    std::swap(psi_old, psi_new);
    /*
    for (i = 1; i <= m; i++) {
      for (j = 1; j <= n; j++) {
        psi[i * (m + 2) + j] = psitmp[i * (m + 2) + j];
      }
    }

    if (!irrotational) {
      for (i = 1; i <= m; i++) {
        for (j = 1; j <= n; j++) {
          zet[i * (m + 2) + j] = zettmp[i * (m + 2) + j];
        }
      }
    }
    */
    nvtxRangePop(); // pop

    if (!irrotational) {
      // update zeta BCs that depend on psi
      boundaryzet(zet, psi, m, n);
    }

    // print loop information

    if (iter % printfreq == 0) {
      if (!checkerr) {
        printf("Completed iteration %d\n", iter);
      } else {
        printf("Completed iteration %d, error = %g\n", iter, error);
      }
    }
  }
  nvtxRangePop(); // pop

  if (iter > numiter)
    iter = numiter;

  tstop = gettime();

  ttot = tstop - tstart;
  titer = ttot / (double)iter;

  // print out some stats

  printf("\n... finished\n");
  printf("After %d iterations, the error is %g\n", iter, error);
  printf("Time for %d iterations was %g seconds\n", iter, ttot);
  printf("Each iteration took %g seconds\n", titer);

  // output results

  HANDLE_ERROR(hipMemcpy(psi, psi_new, sizef, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipPeekAtLastError());

  writedatafiles(psi, m, n, scalefactor);
  writeplotfile(m, n, scalefactor);

  // free un-needed arrays
  HANDLE_ERROR(hipFree(psi_old));
  HANDLE_ERROR(hipFree(psi_new));
  HANDLE_ERROR(hipHostFree(psi));

  if (!irrotational) {
    free(zet);
  }

  printf("... finished\n");

  return 0;
}
